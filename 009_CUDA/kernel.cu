#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "header.h"
#include <stdio.h>
#include <ctime>
__global__ void mullVecsK(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] * b[i];
}

__global__ void addVecK(int *a, unsigned int _s)
{
	int i = threadIdx.x;
	a[i] = a[i] + a[i + _s];
	__syncthreads();
}

void mullWithCuda(int *c, const int *a, const int *b, unsigned int size);
int dotProductWithCuda(const int *a, const int *b, size_t _s);
int addVecWithCuda(int * v, size_t _s);
void __addVecCuda(int * v, size_t _s);

int main()
{
	size_t arraySize = 0;
	int *a,*b,*c;		
	clock_t start, end;
	for (int i = 0; i < 3; i++) {
		if (i == 0)
			arraySize = 512;
		if (i == 1)
			arraySize = 1024;
		if (i == 2)
			arraySize = 131072;
		
		a = new int[arraySize];
		b = new int[arraySize];
		c = new int[arraySize];
		randArr(a, arraySize);
		randArr(b, arraySize);
		
		start = clock();
		int dotProdWC = dotProductWithCuda(a, b, arraySize);
		end = clock();
		auto dur = end - start;
		printf("dot product with CUDA for %d elem = %d time: %d\n", arraySize, dotProdWC, dur);

		start =  clock();
		int dotProd = dotProduct(a, b, arraySize);
		end = clock();
		dur = end - start;
		printf("dot product without CUDA for %d elem= %d time: %d\n", arraySize, dotProd, dur);
		
		delete[] a;
		delete[] b;
		delete[] c;
	}
	return 0;
}

void mullWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));


	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	mullVecsK << <1, size >> > (dev_c, dev_a, dev_b);

	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

void __addVecCuda(int * v, size_t _s)
{
	while (_s != 1) {
		_s /= 2;
		addVecK << <1, _s >> > (v, _s);
	}
}

int addVecWithCuda(int * v, size_t _s)
{
	int * dev_v = 0;
	hipMalloc((void **)&dev_v, _s * sizeof(int));
	hipMemcpy(dev_v, v, _s * sizeof(int), hipMemcpyHostToDevice);
	int * res = new int[_s] {0};
	auto sizes = getSizesOfKernels(_s);
	size_t off = 0;
	for (size_t i = 0; i < sizes.size(); i++)
	{
		if (sizes[i] == 1) {
			off += sizes[i];
			continue;
		}
		__addVecCuda(dev_v + off, sizes[i]);
		hipMemcpy(dev_v + i, dev_v + off, sizeof(int), hipMemcpyDeviceToDevice);
		off += sizes[i];
	}
	if (sizes.size() == 2)
		__addVecCuda(dev_v, 2);
	hipMemcpy(res, dev_v, _s * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_v);
	int result = res[0];
	delete[] res;
	return result;
}

int dotProductWithCuda(const int *a, const int *b, size_t _s)
{
	int * c = new int[_s] {0};
	mullWithCuda(c, a, b, _s);
	int res = addVecWithCuda(c, _s);
	delete[] c;
	return res;
}
